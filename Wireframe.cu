#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <unistd.h>

#include <math.h>

#include <iostream>
#include <vector>
#include <limits>

#include <time.h>
#include <sys/time.h>

#include <X11/Xlib.h>
#include <X11/Xutil.h>
#include <X11/Xatom.h>
#include <X11/keysym.h>
#include <X11/extensions/Xrandr.h>
#include <X11/extensions/Xfixes.h>

using namespace std;

#define Max( n1, n2 )							( ( n1 ) > ( n2 ) ? ( n1 ) : ( n2 ) )
#define Min( n1, n2 )							( ( n1 ) < ( n2 ) ? ( n1 ) : ( n2 ) )

#define Abs( Nn )									( Nn < 0 ? -Nn : Nn )
#define IsNumber( Ch )							( ( Ch >= '0' ) && ( Ch <= '9' ) )

#define PI       3.14159265
#define PVAL	   ( PI / 180 )

#define ALPHA		3
#define RED			2
#define GREEN		1
#define BLUE		0

// --------------------------------------------------------------------- //

typedef struct Float3{
	float X, Y, Z;
}Float3;

typedef struct Face{
	uint32_t f[ 5 ];
}Face;

// --------------------------------------------------------------------- //

struct Canvas{
	uint32_t* Pixels;
	uint32_t* GPUPixels;
	uint32_t  Nx, Ny;

	bool OpenWindow;
	Display *Dsp;
	XSetWindowAttributes Wa;
	XVisualInfo Vi;
	Window Win;
	XImage* Xim;
	GC Gc;
};

// --------------------------------------------------------------------- //

struct Object{
	std::vector<Face>   Faces;
	std::vector<Float3> Verts;
	std::vector<Float3> PerspVerts;

	Float3*  GPUVerts;
	Face*    GPUFaces;

	uint32_t N_Verts, N_Faces;
};

// --------------------------------------------------------------------- //

Canvas   CreateCanvas( uint32_t _Nx, uint32_t _Ny, bool OpenWindow );
void     FreeCanvas( Canvas& Cnv );
void     ClearCanvas( Canvas& Cnv );
void     FlipCanvas( Canvas& Cnv );
void     SaveCanvas( Canvas& Cnv, const char* FileName );

// --------------------------------------------------------------------- //

uint64_t TimeGet();
int      GetInt( FILE* Fl );
Object   Read( char* Path );
void     Print( Object& Obj );

void     DrawLine( Canvas& Cnv, int Px1, int Py1, float Pz1, int Px2, int Py2, float Pz2 );
void     RotateAndDrawObject( Canvas& Cnv, Object& Obj, Float3 Cam, float Rotate );

// --------------------------------------------------------------------- //

__global__ void GPURotateAndDrawObject( Canvas Cnv, Object Obj, Float3 Cam, float Rotate ){
	int Nf = blockIdx.x;
	int Nv = threadIdx.x;

	float Scale, Prop, Sin, Cos;
	Float3 Dif, Persp1, Persp2;
	uint32_t P1, P2;

	float Px, Py, Pz, _Nx, _Ny, _Nz, Dz;
	uint32_t Color, *Pixels = NULL;
	int Dx, Dy, Nn, Nm;
	uint8_t ByteColor;

	if( !Cnv.Nx || !Cnv.Ny )
		return;

	Scale = ( Min( Cnv.Nx, Cnv.Ny ) / 2 );
	Sin = sin( Rotate * PVAL );
	Cos = cos( Rotate * PVAL );

	if( Nf < Obj.N_Faces ){
		if( Nv < 4 ){

			P1 = Obj.GPUFaces[ Nf ].f[ Nv ] - 1;
			P2 = Obj.GPUFaces[ Nf ].f[ Nv + 1 ] - 1;


			Persp1.X = ( Cos * Obj.GPUVerts[ P1 ].X - Sin * Obj.GPUVerts[ P1 ].Z );
			Persp1.Z = ( Sin * Obj.GPUVerts[ P1 ].X + Cos * Obj.GPUVerts[ P1 ].Z );
			Persp1.Y = -Obj.GPUVerts[ P1 ].Y;

			Prop = Abs( Cam.Z - Persp1.Z ) * 0.5;
			Prop = ( Prop == 0 ? 0.0001 : Prop );

			Dif.X = Persp1.X - Cam.X;
			Dif.Y = Persp1.Y - Cam.Y;

			Persp1.X = (float)( Cnv.Nx / 2.0 ) + ( Dif.X * Scale / Prop );
			Persp1.Y = (float)( Cnv.Ny / 2.0 ) + ( Dif.Y * Scale / Prop );


			Persp2.X = ( Cos * Obj.GPUVerts[ P2 ].X - Sin * Obj.GPUVerts[ P2 ].Z );
			Persp2.Z = ( Sin * Obj.GPUVerts[ P2 ].X + Cos * Obj.GPUVerts[ P2 ].Z );
			Persp2.Y = -Obj.GPUVerts[ P2 ].Y;

			Prop = Abs( Cam.Z - Persp2.Z ) * 0.5;
			Prop = ( Prop == 0 ? 0.0001 : Prop );

			Dif.X = Persp2.X - Cam.X;
			Dif.Y = Persp2.Y - Cam.Y;

			Persp2.X = (float)( Cnv.Nx / 2.0 ) + ( Dif.X * Scale / Prop );
			Persp2.Y = (float)( Cnv.Ny / 2.0 ) + ( Dif.Y * Scale / Prop );


			Persp1.Z = Max( Min( Persp1.Z, 1.0 ), -1.0 );
			Persp2.Z = Max( Min( Persp2.Z, 1.0 ), -1.0 );


			Dx = Persp2.X - Persp1.X;
			Dy = Persp2.Y - Persp1.Y;
			Dz = Persp2.Z - Persp1.Z;

			Nn = Max( Abs( Dx ), Abs( Dy ) );
			if( !Nn )
				return;

			Px = (float)Dx / (float)Nn;
			Py = (float)Dy / (float)Nn;
			Pz = (float)Dz / (float)Nn;

			_Nx = Persp1.X;
			_Ny = Persp1.Y;
			_Nz = Persp1.Z;

			for( Nm = 0 ; Nm <= Nn ; Nm++ ){
				ByteColor = (int)( 255 * ( ( _Nz + 1.0 ) / 2.0 ) );
				Color = ( ByteColor << 16 ) | ( ByteColor << 8 ) | ByteColor;

				if( ( _Nx >= 0 ) && ( _Nx < Cnv.Nx ) && ( _Ny >= 0 ) && ( _Ny < Cnv.Ny ) ){
					Pixels = &Cnv.GPUPixels[ (int)_Nx + (int)_Ny * Cnv.Nx ];
					atomicMax( Pixels, Color );
				}
				_Nx += Px;
				_Ny += Py;
				_Nz += Pz;
			}
		}
	}
}

// --------------------------------------------------------------------- //

Canvas CreateCanvas( uint32_t _Nx, uint32_t _Ny, bool OpenWindow = false ){
	Canvas Cnv;
	int Nw, Nh;
	uint64_t Att;
	uint32_t Ns;

	memset( (void*)&Cnv, 0, sizeof( Canvas ) );

	if( !_Nx || !_Ny )
		_Nx = _Ny = 400;

	Cnv.Pixels = (uint32_t*)malloc( 4 * _Nx * _Ny );
	if( !Cnv.Pixels ){
		puts( "Falha ao alocar memória!!" );
		exit( 0 );
	}

  hipMalloc( (void **)&Cnv.GPUPixels, 4 * _Nx * _Ny );

	Cnv.Nx = _Nx;
	Cnv.Ny = _Ny;

	ClearCanvas( Cnv );

	if( OpenWindow == true ){

		Cnv.Dsp = XOpenDisplay( NULL );
		if( !Cnv.Dsp ){
			puts( "ERRO: Problema ao abrir display da janela!" );
			return( Cnv );
		}

		Cnv.Wa.background_pixel = 0;
		Cnv.Wa.override_redirect = 1;

		Att = CWBackPixel | CWColormap;
		Ns = DefaultScreen( Cnv.Dsp );

		if( !XMatchVisualInfo( Cnv.Dsp, Ns, 24, TrueColor, &Cnv.Vi ) ){
			puts( "ERRO: Problema ao setar configuração da janela!" );
			return( Cnv );
		}

		Nw = ( DisplayWidth( Cnv.Dsp, Ns ) / 2 ) - (int)( _Nx / 2 );
		Nh = ( DisplayHeight( Cnv.Dsp, Ns ) / 2 ) - (int)( _Ny / 2 );
		Cnv.Win = XCreateWindow( Cnv.Dsp, RootWindow( Cnv.Dsp, Ns ), Nw, Nh, _Nx, _Ny, 0, Cnv.Vi.depth, InputOutput, Cnv.Vi.visual, Att, &Cnv.Wa );
		if( !Cnv.Win ){
			puts( "ERRO: Problema ao abrir janela!" );
			return( Cnv );
		}

		XSelectInput( Cnv.Dsp, Cnv.Win, ExposureMask | KeyPressMask | KeyReleaseMask | ButtonPressMask | ButtonReleaseMask );
		XStoreName( Cnv.Dsp, Cnv.Win, "Janelinha" );

		Cnv.Gc = XCreateGC( Cnv.Dsp, Cnv.Win, 0, NULL );
		XSetForeground( Cnv.Dsp, Cnv.Gc, 0 );

		Atom At = XInternAtom( Cnv.Dsp , "_NET_WM_STATE", 0 );
		XSetWMProtocols( Cnv.Dsp, Cnv.Win, &At, 1 );
		XMapWindow( Cnv.Dsp, Cnv.Win );

		if( Cnv.Xim ){
			XDestroyImage( Cnv.Xim );
			Cnv.Xim = NULL;
		}

		XResizeWindow( Cnv.Dsp, Cnv.Win, _Nx, _Ny );

		Cnv.Xim = XCreateImage( Cnv.Dsp, Cnv.Vi.visual, Cnv.Vi.depth, ZPixmap, 0, (char*)Cnv.Pixels, _Nx, _Ny, 32, 0 );
		if( !Cnv.Xim ){
			puts( "ERRO: Problema ao setar imagem!" );
			return( Cnv );
		}

		XFlush( Cnv.Dsp );
		XSync( Cnv.Dsp, 0 );
		XPending( Cnv.Dsp );
		usleep( 200000 );

		Cnv.OpenWindow = true;
	}

	return( Cnv );
}

// --------------------------------------------------------------------- //

void FreeCanvas( Canvas& Cnv ){

	if( Cnv.OpenWindow == true ){
		if( Cnv.Xim ){
			XDestroyImage( Cnv.Xim );
			Cnv.Xim = NULL;
		}
		if( Cnv.Dsp ){
			XCloseDisplay( Cnv.Dsp );
			Cnv.Dsp = NULL;
		}
	}
	else
		if( Cnv.Pixels ){
			free( Cnv.Pixels );
			hipFree( Cnv.GPUPixels );
		}

	memset( (void*)&Cnv, 0, sizeof( Canvas ) );
}

// --------------------------------------------------------------------- //

void ClearCanvas( Canvas& Cnv ){

	if( Cnv.Pixels )
		memset( (void*)Cnv.Pixels, 0, 4 * Cnv.Nx * Cnv.Ny );
	hipMemset( Cnv.GPUPixels, 0, 4 * Cnv.Nx * Cnv.Ny );
}

// --------------------------------------------------------------------- //

void FlipCanvas( Canvas& Cnv ){

	if( !Cnv.Xim || !Cnv.Dsp )
		return;

	XPutImage( Cnv.Dsp, Cnv.Win, Cnv.Gc, Cnv.Xim, 0, 0, 0, 0, Cnv.Nx, Cnv.Ny );
}

// --------------------------------------------------------------------- //

void SaveCanvas( Canvas& Cnv, const char* FileName ){
	uint8_t* Data = NULL;
	FILE* Fl = NULL;
	int Nn;

	if( !FileName ){
		puts( "Sem filename!!" );
		return;
	}

	Fl = fopen( FileName, "w" );
	if( !Fl ){
		puts( "Erro ao abrir arquivo!!" );
		return;
	}

	fprintf( Fl, "P6\n%d %d\n255\n", Cnv.Nx, Cnv.Ny );
	for( Nn = 0, Data = (uint8_t*)Cnv.Pixels ; Nn < Cnv.Nx * Cnv.Ny ; Nn++, Data += 4 ){
		fwrite( Data + RED  , 1, 1, Fl );
		fwrite( Data + GREEN, 1, 1, Fl );
		fwrite( Data + BLUE , 1, 1, Fl );
	}

	fclose( Fl );
}

// --------------------------------------------------------------------- //

uint64_t TimeGet(){
	struct timeval tv;
	uint64_t Clk;

	gettimeofday( &tv, NULL );
	Clk = 1000000 * tv.tv_sec + tv.tv_usec;

	return( Clk );
}

// --------------------------------------------------------------------- //

int GetInt( FILE* Fl ){
	char Ch;
	int Nn;

	if( !Fl )
		return( 0 );

	Nn = 0;

	do{
		Ch = fgetc( Fl );
		if( Ch == '\n' )
			return( 0 );
	}while( !IsNumber( Ch ) && !feof( Fl ) );

	do{
		Nn = ( Nn * 10 ) + ( Ch - '0' );
		Ch = fgetc( Fl );
		if( Ch == '/' ){
			do{
				Ch = fgetc( Fl );
			}while( ( Ch != '\n' ) && ( Ch != ' ' ) && !feof( Fl ) );
		}
	}while( IsNumber( Ch ) && !feof( Fl ) );

	fseek( Fl, -1, SEEK_CUR );

	return( Nn );
}

// --------------------------------------------------------------------- //

Object Read( char* Path ){
	Float3 Vmin, Vmax, Dif;
	FILE* Fl = NULL;
	int Size, Nn, Nm;
	float Mn, Prop;
	Object Obj;
	Float3 Nf;
	char Ch;

	if( Path == NULL ){
		puts( "ERRO: Nome do arquivo nulo!" );
		exit( 0 );
	}

	Fl = fopen( Path, "rb" );
	if( Fl == NULL ){
		puts( "ERRO: Arquivo inexistente!" );
		exit( 0 );
	}

	Mn = 0.0;
	Vmin.X = Vmin.Y = Vmin.Z = std::numeric_limits<float>::max();
	Vmax.X = Vmax.Y = Vmax.Z = std::numeric_limits<float>::min();

	// Lendo vertices e faces
	while( !feof( Fl ) ){
		Ch = fgetc( Fl );

		switch( Ch ){

		case( 'v' ):
			if( ( Ch = fgetc( Fl ) ) != ' ' ){
				while( ( Ch != '\n' ) && !feof( Fl ) )
					Ch = fgetc( Fl );
				break;
			}
			fscanf( Fl, "%f %f %f\n", &Nf.X, &Nf.Y, &Nf.Z );

			Size = Obj.Verts.size();
			Obj.Verts.resize( Size + 1 );
			Obj.Verts[ Size ].X = Nf.X;
			Obj.Verts[ Size ].Y = Nf.Y;
			Obj.Verts[ Size ].Z = Nf.Z;

			Vmin.X = Min( Vmin.X, Nf.X );
			Vmax.X = Max( Vmax.X, Nf.X );

			Vmin.Y = Min( Vmin.Y, Nf.Y );
			Vmax.Y = Max( Vmax.Y, Nf.Y );

			Vmin.Z = Min( Vmin.Z, Nf.Z );
			Vmax.Z = Max( Vmax.Z, Nf.Z );
		break;

		case( 'f' ):
			Size = Obj.Faces.size();
			Obj.Faces.resize( Size + 1 );

			Nm = 0;
			while( ( Nn = GetInt( Fl ) ) != 0 ){
				Obj.Faces[ Size ].f[ Nm ] = Nn;
				Nm++;
			}
			for( Nn = 0 ; Nm < 5 ; Nn++, Nm++ )
				Obj.Faces[ Size ].f[ Nm ] = Obj.Faces[ Size ].f[ Nn ];

		break;

		case( '#' ):
		default:
			while( ( Ch != '\n' ) && !feof( Fl ) )
				Ch = fgetc( Fl );
		break;
		}
	}

	// Alocando vetor

	Obj.N_Verts = Obj.Verts.size();
	Obj.N_Faces = Obj.Faces.size();


	Obj.PerspVerts.resize( Obj.N_Verts );

	// Centralizando imagem
	Dif.X = Vmin.X + ( ( Vmax.X - Vmin.X ) / 2 );
	Dif.Y = Vmin.Y + ( ( Vmax.Y - Vmin.Y ) / 2 );
	Dif.Z = Vmin.Z + ( ( Vmax.Z - Vmin.Z ) / 2 );

	Mn = 0;
	for( Nn = 0 ; Nn < (int)Obj.N_Verts ; Nn++ ){
		Obj.Verts[ Nn ].X -= Dif.X;
		Obj.Verts[ Nn ].Y -= Dif.Y;
		Obj.Verts[ Nn ].Z -= Dif.Z;

		Mn = ( Mn > Abs( Obj.Verts[ Nn ].X ) ? Mn : Obj.Verts[ Nn ].X );
		Mn = ( Mn > Abs( Obj.Verts[ Nn ].Y ) ? Mn : Obj.Verts[ Nn ].Y );
		Mn = ( Mn > Abs( Obj.Verts[ Nn ].Z ) ? Mn : Obj.Verts[ Nn ].Z );
	}

	// Redimensionando imagem para o intervalo [-1.0, 1.0]
	Prop = 1.0 / Mn;
	for( Nn = 0 ; Nn < (int)Obj.N_Verts ; Nn++ ){
		Obj.Verts[ Nn ].X *= Prop;
		Obj.Verts[ Nn ].Y *= Prop;
		Obj.Verts[ Nn ].Z *= Prop;
	}

	// Gerando alocações da GPU

  hipMalloc( (void **)&Obj.GPUFaces, Obj.N_Faces * sizeof( Face ) );
  hipMalloc( (void **)&Obj.GPUVerts, Obj.N_Verts * sizeof( Float3 ) );
	if( !Obj.GPUFaces || !Obj.GPUVerts ){
		puts( "Falha de alocação!!" );
		exit( 0 );
	}

	hipMemcpy( Obj.GPUFaces, &Obj.Faces[ 0 ], Obj.N_Faces * sizeof( Face ), hipMemcpyHostToDevice );
	hipMemcpy( Obj.GPUVerts, &Obj.Verts[ 0 ], Obj.N_Verts * sizeof( Float3 ), hipMemcpyHostToDevice );

	return( Obj );
}

// --------------------------------------------------------------------- //

void Print( Object& Obj ){
	uint32_t Nn, Nm;

	for( Nn = 0 ; Nn < Obj.N_Verts ; Nn++ )
		printf( "Point %d: %f %f %f\n", Nn, Obj.Verts[ Nn ].X, Obj.Verts[ Nn ].Y, Obj.Verts[ Nn ].Z );

	for( Nn = 0 ; Nn < Obj.N_Faces ; Nn++ ){
		printf( "Face %d: ", Nn );
		for( Nm = 0 ; Obj.Faces[ Nn ].f[ Nm ] ; Nm++ )
			printf( "%d ", Obj.Faces[ Nn ].f[ Nm ] - 1 );
		printf( "\n" );
	}
}

// --------------------------------------------------------------------- //

void DrawLine( Canvas& Cnv, int Px1, int Py1, float Pz1, int Px2, int Py2, float Pz2 ){
	float Px, Py, Pz, _Nx, _Ny, _Nz, Dz;
	uint32_t Color, *Pixels = NULL;
	int Dx, Dy, Nn, Nm;
	uint8_t ByteColor;

	if( !Cnv.Pixels )
		return;

	Pz1 = Max( Min( Pz1, 1.0 ), -1.0 );
	Pz2 = Max( Min( Pz2, 1.0 ), -1.0 );

	Dx = Px2 - Px1;
	Dy = Py2 - Py1;
	Dz = Pz2 - Pz1;

	Nn = Max( Abs( Dx ), Abs( Dy ) );
	if( !Nn )
		return;

	Px = (float)Dx / (float)Nn;
	Py = (float)Dy / (float)Nn;
	Pz = (float)Dz / (float)Nn;

	_Nx = Px1;
	_Ny = Py1;
	_Nz = Pz1;

	for( Nm = 0 ; Nm <= Nn ; Nm++ ){
		ByteColor = (int)( 255 * ( ( _Nz + 1.0 ) / 2.0 ) );
		Color = ( ByteColor << 16 ) | ( ByteColor << 8 ) | ByteColor;

		if( ( _Nx >= 0 ) && ( _Nx < Cnv.Nx ) && ( _Ny >= 0 ) && ( _Ny < Cnv.Ny ) ){
			Pixels = &Cnv.Pixels[ (int)_Nx + (int)_Ny * Cnv.Nx ];
			*Pixels = Max( Color, *Pixels );
		}
		_Nx += Px;
		_Ny += Py;
		_Nz += Pz;
	}
}

// --------------------------------------------------------------------- //

void RotateAndDrawObject( Canvas& Cnv, Object& Obj, Float3 Cam, float Rotate ){
	float Scale, Prop, Sin, Cos;
	uint32_t P1, P2;
	Float3 Dif;
	int Nn, Nm;

	if( !Cnv.Nx || !Cnv.Ny )
		return;

	Scale = ( Min( Cnv.Nx, Cnv.Ny ) / 2 );
	Sin = sin( Rotate * PVAL );
	Cos = cos( Rotate * PVAL );

	for( Nn = 0 ; Nn < (int)Obj.N_Verts ; Nn++ ){

		Obj.PerspVerts[ Nn ].X = ( Cos * Obj.Verts[ Nn ].X - Sin * Obj.Verts[ Nn ].Z );
		Obj.PerspVerts[ Nn ].Z = ( Sin * Obj.Verts[ Nn ].X + Cos * Obj.Verts[ Nn ].Z );
		Obj.PerspVerts[ Nn ].Y = -Obj.Verts[ Nn ].Y;

		Prop = Abs( Cam.Z - Obj.PerspVerts[ Nn ].Z ) * 0.5;
		Prop = ( Prop == 0 ? 0.0001 : Prop );
		//printf( "Proportion: %f = %f - %f\n", Prop, Cam.Z, Verts[ Nn ].X );

		Dif.X = Obj.PerspVerts[ Nn ].X - Cam.X;
		Dif.Y = Obj.PerspVerts[ Nn ].Y - Cam.Y;

		Obj.PerspVerts[ Nn ].X = (float)( Cnv.Nx / 2.0 ) + ( Dif.X * Scale / Prop );
		Obj.PerspVerts[ Nn ].Y = (float)( Cnv.Ny / 2.0 ) + ( Dif.Y * Scale / Prop );
	}

	for( Nn = 0 ; Nn < (int)Obj.N_Faces ; Nn++ ){
		for( Nm = 0 ; Nm < 4 ; Nm++ ){
			P1 = Obj.Faces[ Nn ].f[ Nm ] - 1;
			P2 = Obj.Faces[ Nn ].f[ Nm + 1 ] - 1;
			DrawLine( Cnv, Obj.PerspVerts[ P1 ].X, Obj.PerspVerts[ P1 ].Y, Obj.PerspVerts[ P1 ].Z,
                     Obj.PerspVerts[ P2 ].X, Obj.PerspVerts[ P2 ].Y, Obj.PerspVerts[ P2 ].Z  );
		}
	}

}

// --------------------------------------------------------------------- //

void Help(){

	puts( "\t-s: Executa o algoritmo sequencial." );
	puts( "\t-f <file>: Especifica o algoritmo de entrada." );
	puts( "\t-r <N>: Especifica a quantidade de posições a serem geradas durante a rotação do objeto(180 por padrão)." );
	puts( "\t-w: Abre uma janela para exibir a projeção criada." );
	puts( "\t-x <N>: Altera a largura da tela(400 por padrão)." );
	puts( "\t-y <N>: Altera a altura da tela(400 por padrão)." );
	puts( "\t-p: Nome da pasta para salvar as imagens geradas." );
	puts( "\t-t: Define o número de vezes que o algoritmo será executado consecutivamente(1 por padrão)." );
	puts( "\t-h: Abre o menu de ajuda." );
}

// --------------------------------------------------------------------- //

int main( int Argc, char** Argv ){
	uint32_t Nrot = 180, Nx = 400, Ny = 400, Tests = 1, Nt;
	char Buff[ 64 ], *FileName = NULL, *Path = NULL;
	uint64_t T1, Total, Med;
	bool Seq, Win;
	Float3 Cam;
	Canvas Cnv;
	Object Obj;
	int Nn;

	if( Argc <= 1 ){
		puts( "Faltam argumentos:" );
		Help();

		return( 0 );
	}

	Seq = Win = false;

	for( Nn = 1 ; Nn < Argc ; Nn++ ){
		if( !strcmp( Argv[ Nn ], "-s" ) )
			Seq = true;

		if( !strcmp( Argv[ Nn ], "-w" ) )
			Win = true;

		if( !strcmp( Argv[ Nn ], "-h" ) ){
			Help();
			return( 0 );
		}

		if( ( !strcmp( Argv[ Nn ], "-f" ) ) && ( Argv[ Nn + 1 ] != NULL ) ){
			Nn++;
			FileName = Argv[ Nn ];
		}

		if( ( !strcmp( Argv[ Nn ], "-r" ) ) && ( Argv[ Nn + 1 ] != NULL ) ){
			Nn++;
			Nrot = atoi( Argv[ Nn ] );
		}

		if( ( !strcmp( Argv[ Nn ], "-x" ) ) && ( Argv[ Nn + 1 ] != NULL ) ){
			Nn++;
			Nx = atoi( Argv[ Nn ] );
		}

		if( ( !strcmp( Argv[ Nn ], "-y" ) ) && ( Argv[ Nn + 1 ] != NULL ) ){
			Nn++;
			Ny = atoi( Argv[ Nn ] );
		}

		if( ( !strcmp( Argv[ Nn ], "-t" ) ) && ( Argv[ Nn + 1 ] != NULL ) ){
			Nn++;
			Tests = atoi( Argv[ Nn ] );
		}

		if( ( !strcmp( Argv[ Nn ], "-p" ) ) && ( Argv[ Nn + 1 ] != NULL ) ){
			Nn++;
			Path = Argv[ Nn ];
		}
	}

	Cam.X = Cam.Y = 0;
	Cam.Z = 3;

	Obj = Read( FileName );

	printf( "%s: %u vertices e %u faces %s(%d posições).\n", FileName, Obj.N_Verts, Obj.N_Faces, Seq == true ? "sequencial" : "em paralelo", Nrot );
	//Print( Obj );

	Cnv = CreateCanvas( Nx, Ny, Win );

	Med = 0;
	for( Nt = 0 ; Nt < Tests ; Nt++ ){
		Total = 0;
		for( Nn = 0 ; Nn < Nrot ; Nn++ ){
			ClearCanvas( Cnv );

			if( Seq == true ){
				T1 = TimeGet();
				RotateAndDrawObject( Cnv, Obj, Cam, Nn * ( 360.0 / Nrot ) );
				Total += ( TimeGet() - T1 );
			}
			else{
				T1 = TimeGet();
				GPURotateAndDrawObject <<< Obj.N_Faces, 4 >>> ( Cnv, Obj, Cam, Nn * ( 360.0 / Nrot ) );
				hipMemcpy( Cnv.Pixels, Cnv.GPUPixels, Cnv.Nx * Cnv.Ny * 4, hipMemcpyDeviceToHost);		
				Total += ( TimeGet() - T1 );
			}

			if( Path != NULL ){
				sprintf( Buff, "%s/Image%d.ppm", Path, Nn );
				SaveCanvas( Cnv, Buff );
			}
			if( Win == true ){
				FlipCanvas( Cnv );
				usleep( 10000 );
			}
		}

		printf( "Tempo total gasto com processamento: %lu.%06lus\n", Total / 1000000, Total % 1000000 );
		Med += Total;
	}

	Med /= Tests;
	printf( "Tempo medio: %lu.%06lus\n", Med / 1000000, Med % 1000000 );

	FreeCanvas( Cnv );

	return( 0 );
}

// --------------------------------------------------------------------- //
